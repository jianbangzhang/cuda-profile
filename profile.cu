#include <iostream>
#include <iomanip>
#include <hip/hip_runtime.h>
#include <hipblas.h>

#define CUDA_CHECK(call) \
    do { \
        hipError_t error = call; \
        if (error != hipSuccess) { \
            std::cerr << "CUDA error: " << hipGetErrorString(error) << std::endl; \
            exit(1); \
        } \
    } while(0)

// 根据GPU架构估算理论峰值性能（GFLOPS）
double getTheoreticalGFLOPS(int major, int minor, int multiProcessorCount, int clockRate) {
    double gflops = 0.0;
    
    // CUDA核心数估算（每个SM的核心数）
    int coresPerSM = 0;
    
    if (major == 2) {
        coresPerSM = 32;  // Fermi
    } else if (major == 3) {
        coresPerSM = 192; // Kepler
    } else if (major == 5) {
        coresPerSM = 128; // Maxwell
    } else if (major == 6) {
        if (minor == 0) coresPerSM = 64;   // Pascal GP100
        else coresPerSM = 128;             // Pascal GP10x
    } else if (major == 7) {
        if (minor == 0) coresPerSM = 64;   // Volta V100
        else coresPerSM = 64;              // Turing
    } else if (major == 8) {
        if (minor == 0) coresPerSM = 64;   // Ampere A100
        else coresPerSM = 128;             // Ampere GA10x
    } else if (major == 9) {
        coresPerSM = 128; // Ada Lovelace / Hopper (估算)
    } else {
        coresPerSM = 64;  // 默认估算
    }
    
    int totalCores = coresPerSM * multiProcessorCount;
    // 理论峰值 = 核心数 × 时钟频率 × 2 (FMA指令可以同时做乘法和加法)
    gflops = (totalCores * clockRate * 2.0) / 1000000.0; // 转换为GFLOPS
    
    return gflops;
}

// 计算内存带宽 (GB/s)
double getMemoryBandwidth(int memoryClockRate, int memoryBusWidth) {
    // 内存带宽 = 内存时钟频率 × 总线宽度 × 2(DDR) / 8(位到字节)
    return (memoryClockRate * 2.0 * memoryBusWidth) / (8.0 * 1000.0);
}

// 计算算术强度的界限点
double calculateBoundaryArithmeticIntensity(double peakGFLOPS, double memoryBandwidthGB) {
    // 界限点：算术强度 = 峰值计算性能 / 内存带宽
    return peakGFLOPS / memoryBandwidthGB;
}

int main() {
    std::cout << "=== GPU Memory Bound vs Computing Bound 分析 ===\n\n";
    
    int deviceCount;
    CUDA_CHECK(hipGetDeviceCount(&deviceCount));
    
    if (deviceCount == 0) {
        std::cout << "没有发现CUDA设备！\n";
        return 1;
    }
    
    for (int device = 0; device < deviceCount; ++device) {
        hipDeviceProp_t prop;
        CUDA_CHECK(hipGetDeviceProperties(&prop, device));
        
        std::cout << "=== 设备 " << device << ": " << prop.name << " ===\n";
        
        // 基本硬件信息
        std::cout << "\n硬件规格:\n";
        std::cout << "  计算能力: " << prop.major << "." << prop.minor << "\n";
        std::cout << "  流多处理器(SM)数量: " << prop.multiProcessorCount << "\n";
        std::cout << "  基础时钟频率: " << prop.clockRate / 1000 << " MHz\n";
        std::cout << "  内存时钟频率: " << prop.memoryClockRate / 1000 << " MHz\n";
        std::cout << "  内存总线宽度: " << prop.memoryBusWidth << " bit\n";
        std::cout << "  全局内存大小: " << prop.totalGlobalMem / (1024*1024*1024) << " GB\n";
        std::cout << "  共享内存大小: " << prop.sharedMemPerBlock / 1024 << " KB per block\n";
        
        // 计算理论峰值性能
        double theoreticalGFLOPS = getTheoreticalGFLOPS(prop.major, prop.minor, 
                                                       prop.multiProcessorCount, 
                                                       prop.clockRate);
        
        // 计算内存带宽
        double memoryBandwidth = getMemoryBandwidth(prop.memoryClockRate, prop.memoryBusWidth);
        
        // 计算界限点
        double boundaryAI = calculateBoundaryArithmeticIntensity(theoreticalGFLOPS, memoryBandwidth);
        
        std::cout << "\n性能分析:\n";
        std::cout << std::fixed << std::setprecision(2);
        std::cout << "  理论峰值计算性能: " << theoreticalGFLOPS << " GFLOPS\n";
        std::cout << "  理论峰值内存带宽: " << memoryBandwidth << " GB/s\n";
        std::cout << "  界限算术强度: " << boundaryAI << " FLOP/Byte\n";
        
        std::cout << "\nMemory Bound vs Computing Bound 分析:\n";
        std::cout << "  当算术强度 < " << boundaryAI << " FLOP/Byte 时: Memory Bound\n";
        std::cout << "  当算术强度 > " << boundaryAI << " FLOP/Byte 时: Computing Bound\n";
        
        // 实际应用示例
        std::cout << "\n常见操作的算术强度参考:\n";
        std::cout << "  向量加法 (C = A + B): ~0.33 FLOP/Byte";
        if (0.33 < boundaryAI) std::cout << " → Memory Bound\n";
        else std::cout << " → Computing Bound\n";
        
        std::cout << "  矩阵-向量乘法: ~2 FLOP/Byte";
        if (2.0 < boundaryAI) std::cout << " → Memory Bound\n";
        else std::cout << " → Computing Bound\n";
        
        std::cout << "  小矩阵乘法 (64x64): ~8 FLOP/Byte";
        if (8.0 < boundaryAI) std::cout << " → Memory Bound\n";
        else std::cout << " → Computing Bound\n";
        
        std::cout << "  大矩阵乘法 (1024x1024+): ~40+ FLOP/Byte";
        if (40.0 < boundaryAI) std::cout << " → Memory Bound\n";
        else std::cout << " → Computing Bound\n";
        
        std::cout << "  卷积 (大kernel): ~20-100 FLOP/Byte";
        if (60.0 < boundaryAI) std::cout << " → Memory Bound\n";
        else std::cout << " → Computing Bound\n";
        
        // 优化建议
        std::cout << "\n优化建议:\n";
        std::cout << "  Memory Bound操作优化策略:\n";
        std::cout << "    - 减少内存访问次数\n";
        std::cout << "    - 使用合并内存访问\n";
        std::cout << "    - 利用共享内存和缓存\n";
        std::cout << "    - 数据重用\n";
        
        std::cout << "  Computing Bound操作优化策略:\n";
        std::cout << "    - 增加并行度\n";
        std::cout << "    - 使用Tensor Cores (如果支持)\n";
        std::cout << "    - 优化算法复杂度\n";
        std::cout << "    - 使用更高效的数学库\n";
        
        // Roofline模型绘制数据
        std::cout << "\nRoofline模型关键点:\n";
        std::cout << "  峰值性能线: " << theoreticalGFLOPS << " GFLOPS (水平线)\n";
        std::cout << "  内存带宽线: Performance = " << memoryBandwidth << " × AI (斜率=" << memoryBandwidth << ")\n";
        std::cout << "  交点 (Ridge Point): (" << boundaryAI << ", " << theoreticalGFLOPS << ")\n";
        
        if (device < deviceCount - 1) {
            std::cout << "\n" << std::string(60, '=') << "\n";
        }
    }
    
    return 0;
}
